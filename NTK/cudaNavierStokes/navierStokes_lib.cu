#include "hip/hip_runtime.h"
#include "functions.h"
#include <GL/gl.h>
#include <GLFW/glfw3.h>
#include <cmath>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <cmath>


using namespace std;

const char *SNAPSHOT_FILENAME = "snapshots.txt";

const bool implicit = true; // true for implicit, false for explicit

// mainfluids_visualization.cu
#include "obstacles.h"

// Viridis colormap data (You need to include all 256 entries)
// For brevity, only a few entries are shown here. Include all in your code.

// Global variables
Vector2f C;
Vector2f F;
// Vector2f C1;
// Vector2f F1;
float global_decay_rate = DECAY_RATE;

// Function to decay the force
void decayForce() {
    float nx = F.x - global_decay_rate;
    float ny = F.y - global_decay_rate;
    nx = (nx > 0) ? nx : 0.0f;
    ny = (ny > 0) ? ny : 0.0f;
    F = Vector2f(nx, ny);
}

// First step: apply the external force field to the data
__device__ void force(Vector2f x, Vector2f* field, Vector2f C, Vector2f F, float timestep, float r, unsigned dim) {
    if (periodic == 1){
        // Apply periodic wrapping to the position
        // Apply periodic wrapping to the center position
        Vector2f xC = x - C;
        xC.x = fmodf(xC.x + dim / 2.0f, dim) - dim / 2.0f;
        xC.y = fmodf(xC.y + dim / 2.0f, dim) - dim / 2.0f;

        float exp_val = (xC.x * xC.x + xC.y * xC.y) / r;
        float factor = timestep * expf(-exp_val) * 0.001f;
        Vector2f temp = F * factor;

        int idx = IND(static_cast<int>(x.x), static_cast<int>(x.y), dim);
        field[idx] += temp;
    }

    else {
        float xC[2] = { x.x - C.x, x.y - C.y };
        float exp_val = (xC[0] * xC[0] + xC[1] * xC[1]) / r;
        int i = static_cast<int>(x.x);
        int j = static_cast<int>(x.y);
        float factor = timestep * expf(-exp_val) * 0.001f;
        Vector2f temp = F * factor;
        if (i >= 0 && i < dim && j >= 0 && j < dim) {
            field[IND(i, j, dim)] += temp;
        }
    }
}

// Bilinear interpolation
/***
If the interpolated position falls outside the simulation grid, the function returns a zero vector,
imposing a zero velocity at the boundaries.
*/

// Second step: advect the data through method of characteristics
__device__ void advect(Vector2f x, Vector2f* field, Vector2f* velfield, int* obstacleField, float timestep, float rdx, unsigned dim) {
    float dt0 = timestep * rdx;

    // Compute k1
    Vector2f k1 = velocityAt(x, velfield, dim);
    Vector2f x1 = x - 0.5f * dt0 * k1;

    // Compute k2
    Vector2f k2 = velocityAt(x1, velfield, dim);
    Vector2f x2 = x - 0.5f * dt0 * k2;

    // Compute k3
    Vector2f k3 = velocityAt(x2, velfield, dim);
    Vector2f x3 = x - dt0 * k3;

    // Compute k4
    Vector2f k4 = velocityAt(x3, velfield, dim);

    // Combine to get final position
    Vector2f pos = x - (dt0 / 6.0f) * (k1 + 2.0f * k2 + 2.0f * k3 + k4);

    if (periodic == 1) {
        // Apply periodic wrapping to the position
        pos.x = fmodf(pos.x + dim, dim);
        pos.y = fmodf(pos.y + dim, dim);
    } else {
        pos.x = fmaxf(0.0f, fminf(pos.x, dim - 1.0f));
        pos.y = fmaxf(0.0f, fminf(pos.y, dim - 1.0f));
    }

    int idx = IND(static_cast<int>(x.x), static_cast<int>(x.y), dim);

    // Check if the backtraced position is inside an obstacle
    int i = static_cast<int>(pos.x);
    int j = static_cast<int>(pos.y);
    int posIdx = IND(i, j, dim);

    if (obstacleField[posIdx] == 1) {
        // Do not advect into the obstacle
        field[idx] = Vector2f::Zero();
    } else {
        // Interpolate the field at the backtraced position
        field[idx] = bilinearInterpolation(pos, field, dim);
    }
}

// Third step: diffuse the data
template <typename T>
__device__ void jacobi(Vector2f x, T* field, T* field0, int* obstacleField, float alpha, float beta, T b, T zero, unsigned dim) {
    int i = (int)x.x;
    int j = (int)x.y;

    int idx = IND(i, j, dim);

    if (obstacleField[idx] == 1) {
        // Inside obstacle, keep the field unchanged
        field[idx] = zero;
        return;
    }

    if (periodic == 1) {
        // Use periodic indexing
        int iL = periodicIndex(i - 1, dim);
        int iR = periodicIndex(i + 1, dim);
        int jB = periodicIndex(j - 1, dim);
        int jT = periodicIndex(j + 1, dim);

        // Neighbor values
        T fL = field[IND(iL, j, dim)];
        T fR = field[IND(iR, j, dim)];
        T fB = field[IND(i, jB, dim)];
        T fT = field[IND(i, jT, dim)];

        // Update the current grid point
        field[idx] = (fL + fR + fB + fT + alpha * b) / beta;
    } else {
        // Handle boundaries
        T fL = (i > 0 && obstacleField[IND(i - 1, j, dim)] == 0) ? field[IND(i - 1, j, dim)] : zero;
        T fR = (i < dim - 1 && obstacleField[IND(i + 1, j, dim)] == 0) ? field[IND(i + 1, j, dim)] : zero;
        T fB = (j > 0 && obstacleField[IND(i, j - 1, dim)] == 0) ? field[IND(i, j - 1, dim)] : zero;
        T fT = (j < dim - 1 && obstacleField[IND(i, j + 1, dim)] == 0) ? field[IND(i, j + 1, dim)] : zero;

        // Update the current grid point
        field[idx] = (fL + fR + fB + fT + alpha * b) / beta;
    }
}

// Compute divergence
__device__ float divergence(Vector2f x, Vector2f* from, int* obstacleField, float halfrdx, unsigned dim) {
    int i = static_cast<int>(x.x);
    int j = static_cast<int>(x.y);
    int idx = IND(i, j, dim);

    if (obstacleField[idx] == 1)
        return 0.0f;

    if (periodic == 1) {
        // Use periodic indexing
        int iL = periodicIndex(i - 1, dim);
        int iR = periodicIndex(i + 1, dim);
        int jB = periodicIndex(j - 1, dim);
        int jT = periodicIndex(j + 1, dim);

        Vector2f wL = from[IND(iL, j, dim)];
        Vector2f wR = from[IND(iR, j, dim)];
        Vector2f wB = from[IND(i, jB, dim)];
        Vector2f wT = from[IND(i, jT, dim)];

        float div = halfrdx * ((wR.x - wL.x) + (wT.y - wB.y));
        return div;
    } else {
        Vector2f wL = (i > 0 && obstacleField[IND(i - 1, j, dim)] == 0) ? from[IND(i - 1, j, dim)] : Vector2f::Zero();
        Vector2f wR = (i < dim - 1 && obstacleField[IND(i + 1, j, dim)] == 0) ? from[IND(i + 1, j, dim)] : Vector2f::Zero();
        Vector2f wB = (j > 0 && obstacleField[IND(i, j - 1, dim)] == 0) ? from[IND(i, j - 1, dim)] : Vector2f::Zero();
        Vector2f wT = (j < dim - 1 && obstacleField[IND(i, j + 1, dim)] == 0) ? from[IND(i, j + 1, dim)] : Vector2f::Zero();

        float div = halfrdx * ((wR.x - wL.x) + (wT.y - wB.y));
        return div;
    }
}
// Obtain the approximate gradient of a scalar field
__device__ Vector2f gradient(Vector2f x, float* p, int* obstacleField, float halfrdx, unsigned dim) {
    int i = static_cast<int>(x.x);
    int j = static_cast<int>(x.y);
    int idx = IND(i, j, dim);

    if (obstacleField[idx] == 1)
        return Vector2f::Zero();

    if (periodic == 1) {
        int iL = periodicIndex(i - 1, dim);
        int iR = periodicIndex(i + 1, dim);
        int jB = periodicIndex(j - 1, dim);
        int jT = periodicIndex(j + 1, dim);

        float pL = p[IND(iL, j, dim)];
        float pR = p[IND(iR, j, dim)];
        float pB = p[IND(i, jB, dim)];
        float pT = p[IND(i, jT, dim)];

        Vector2f grad;
        grad.x = halfrdx * (pR - pL);
        grad.y = halfrdx * (pT - pB);
        return grad;
    } else {
        float pL = (i > 0 && obstacleField[IND(i - 1, j, dim)] == 0) ? p[IND(i - 1, j, dim)] : p[idx];
        float pR = (i < dim - 1 && obstacleField[IND(i + 1, j, dim)] == 0) ? p[IND(i + 1, j, dim)] : p[idx];
        float pB = (j > 0 && obstacleField[IND(i, j - 1, dim)] == 0) ? p[IND(i, j - 1, dim)] : p[idx];
        float pT = (j < dim - 1 && obstacleField[IND(i, j + 1, dim)] == 0) ? p[IND(i, j + 1, dim)] : p[idx];

        Vector2f grad;
        grad.x = halfrdx * (pR - pL);
        grad.y = halfrdx * (pT - pB);
        return grad;
    }
}

// Navier-Stokes kernel
__global__ void NSkernel(Vector2f* u, float* p, float* c, int* obstacleField, float c_ambient, float gravity, float betabouyancy, float rdx, float viscosity, Vector2f C, Vector2f F, float timestep, float r, unsigned dim) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int j = blockDim.y * blockIdx.y + threadIdx.y;

    int boolvortex = VORTEX;
    int booljet = FLUID_INJ;

    if (i >= dim || j >= dim)
        return;

    int idx = IND(i, j, dim);

    // Check if the cell is inside the obstacle
    if (obstacleField[idx] == 1) {
        // Set velocity to zero inside the obstacle
        u[idx] = Vector2f::Zero();
        return;
    }

    Vector2f x(static_cast<float>(i), static_cast<float>(j));

    // Force application
    force(x, u, C, F, timestep, r, dim);
    __syncthreads();

    // Advection
    advect(x, u, u, obstacleField, timestep, rdx, dim);
    __syncthreads();

    // Diffusion
    float alpha = rdx * rdx / (viscosity * timestep);
    float beta = 4.0f + alpha;
    for (int iter = 0; iter < NUM_OF_DIFFUSION_STEPS; iter++) {
        jacobi<Vector2f>(x, u, u, obstacleField, alpha, beta, u[idx], Vector2f::Zero(), dim);
        __syncthreads();
    }

    // Pressure calculation
    alpha = -rdx * rdx;
    beta = 4.0f;
    float div = divergence(x, u, obstacleField, 0.5f * rdx, dim);
    jacobi<float>(x, p, p, obstacleField, alpha, beta, div, 0.0f, dim);
    __syncthreads();

    // Pressure gradient subtraction
    Vector2f grad_p = gradient(x, p, obstacleField, 0.5f * rdx, dim);
    u[idx] -= grad_p;
    __syncthreads();

    if (booljet == 1)
        injectFluid(u, dim);
        __syncthreads();

        if (boolvortex == 1)
            applyVortex(u, F,dim);
            __syncthreads();

    if (advect_scalar_bool == 1) {
        // Advection of scalar field c
        advectScalar(x, c, u, obstacleField, timestep, rdx, dim);
        __syncthreads();

    //     // Diffusion of scalar field c
        diffuseScalar(x, c, obstacleField, diffusion_rate, timestep, rdx, dim);
        __syncthreads();

    //     // Apply buoyancy force based on c
        applyBuoyancy(x, u, c, obstacleField, c_ambient, betabouyancy, gravity, dim);
        __syncthreads();
    }
}




