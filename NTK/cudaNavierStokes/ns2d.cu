#include "hip/hip_runtime.h"
#include "navierStokes_lib.cu"
#include <pybind11/numpy.h>
#include <pybind11/pybind11.h>
#include <pybind11/stl.h>
#include <algorithm> // for std::min/max
#include <vector>
#include <array>

namespace py = pybind11;

// Variabili globali
int framecount = 0;
float t_current = 0;
float eps = 0.0f; // per le condizioni al contorno
int shift =10;    // shift delle BC verso l'interno o l'esterno

Vector2f *u, *dev_u;
float *p, *c, *dev_p, *dev_c;
int *obstacleField, *dev_obstacleField;
float M_PI = 3.1415926535f;
bool wantWrite = false; // se true, scrive snapshot su file

float x_min = -1.0f, x_max = 1.0f;
float y_min = -1.0f, y_max = 1.0f;
float dx = (x_max - x_min) / (dim - 1);
float dy = (y_max - y_min) / (dim - 1);
vector<vector<float>>
    snapshots; // ogni snapshot: [ux(0), ..., ux(N), uy(0), ..., uy(N)]
// In cima al file, array globale per la storia delle BC
std::vector<std::vector<std::array<float, 5>>> left_bc_history;
std::vector<std::vector<std::array<float, 5>>> right_bc_history;
std::vector<std::vector<std::array<float, 5>>> top_bc_history;
std::vector<std::vector<std::array<float, 5>>> bottom_bc_history;

void setupNS2d() {

  // Alloca host e device arrays
  obstacleField = (int *)malloc(dim * dim * sizeof(int));
  initializeObstacle(obstacleField, dim, obstacleCenterX, obstacleCenterY, obstacleRadius);

  u = (Vector2f *)malloc(dim * dim * sizeof(Vector2f));
  p = (float*)malloc(dim * dim * sizeof(float));
  c = (float*)malloc(dim * dim * sizeof(float));

  hipMalloc(&dev_u, dim * dim * sizeof(Vector2f));
  hipMalloc(&dev_p, dim * dim * sizeof(float));
  hipMalloc(&dev_c, dim * dim * sizeof(float));
  hipMalloc((void**)&dev_obstacleField, dim * dim * sizeof(int));

  // Inizializza condizioni iniziali
  for (unsigned i = 0; i < dim; i++) {
    float x = x_min + i * dx;
    for (unsigned j = 0; j < dim; j++) {
      float y = y_min + j * dy;
      unsigned idx = i * dim + j;
      u[idx].x =sinf(M_PI * x) * cosf(M_PI * y);
      u[idx].y = cosf(M_PI * x) * sinf(M_PI * y);
    }
  }
  for (unsigned i = 0; i < dim * dim; i++) {
    p[i] = 0.0f;
    c[i] = 0.0f;
    // obstacleField[i] = 0; // Inizializza il campo degli ostacoli a 0
  }

  hipMemcpy(dev_u, u, dim * dim * sizeof(Vector2f), hipMemcpyHostToDevice);
  hipMemcpy(dev_p, p, dim * dim * sizeof(float),
             hipMemcpyHostToDevice);
  hipMemcpy(dev_c, c, dim * dim * sizeof(float),
             hipMemcpyHostToDevice);
              // Copy obstacle field to device
  hipMemcpy(dev_obstacleField, obstacleField, dim * dim * sizeof(int), hipMemcpyHostToDevice);

  framecount = 0;
  // Se vuoi, svuota anche snapshots se ne tieni traccia
}
// --- BC su x = 0 ---
std::vector<std::array<float, 5>> get_left_bc() {
  std::vector<std::array<float, 5>> bc(dim);

  // Calcola l'indice più vicino alla posizione eps
  // int i = int(std::round(eps / dx)); // i=0 (bordo), i=1,2... (più dentro)
  int i = shift;
  float x_val = x_min + i * dx;

  for (int j = 0; j < dim; ++j) {
    float y_val = y_min + j * dy;
    int idx = i * dim + j;
    bc[j] = {t_current, x_val, y_val, u[idx].x, u[idx].y};
  }
  return bc;
}

std::vector<std::array<float, 5>> get_right_bc() {
  std::vector<std::array<float, 5>> bc(dim);

  int i = (dim - 1) - shift; // x=max per shift=0, più interno per shift>0
  float x_val = x_min + i * dx;

  for (int j = 0; j < dim; ++j) {
    float y = y_min + j * dy;
    int idx = i * dim + j; // riga i, colonna j
    bc[j] = {t_current, x_val, y, u[idx].x, u[idx].y};
  }
  return bc;
}

std::vector<std::array<float, 5>> get_top_bc() {
  std::vector<std::array<float, 5>> bc(dim);
  int j = (dim - 1) - shift; // y=max per shift=0, più interno per shift>0
  float y_val = y_min + j * dy;
  for (int i = 0; i < dim; ++i) {
    float x = x_min + i * dx;
    int idx = i * dim + j; // riga i, colonna j
    bc[i] = {t_current, x, y_val, u[idx].x, u[idx].y};
  }
  return bc;
}

std::vector<std::array<float, 5>> get_bottom_bc() {
  std::vector<std::array<float, 5>> bc(dim);
  int j = shift; // y=0 per shift=0, più interno per shift>0
  for (int i = 0; i < dim; ++i) {
    float x = x_min + i * dx;
    int idx = i * dim + j; // y=0
    bc[i] = {t_current, x, y_min, u[idx].x, u[idx].y};
  }
  return bc;
}
void mainNS2d() {
  printf("Starting 2D Navier-Stokes simulation...\n");
  //generate a random float number

  float betabouyancy = BETA_BOUYANCY; // Buoyancy coefficient
  float gravity = -9.81f;

  // Obstacle parameters
  // float obstacleCenterX = dim / 2.0f; // Center of the domain
  // float obstacleCenterY = dim / 2.0f;
  // float obstacleRadius = dim / 10.0f; // Adjust as needed
  // CUDA grid and block dimensions
  dim3 threads(BLOCKSIZEX, BLOCKSIZEY);
  dim3 blocks((dim + BLOCKSIZEX - 1) / BLOCKSIZEX, (dim + BLOCKSIZEY - 1) / BLOCKSIZEY);
  // initializeCylinder(obstacleField, dim, obstacleCenterX, obstacleCenterY, obstacleRadius);
  initializeObstacle(obstacleField, dim, obstacleCenterX, obstacleCenterY, obstacleRadius);
  hipMemcpy(dev_obstacleField, obstacleField, dim * dim * sizeof(int), hipMemcpyHostToDevice);

  // ------------------------------------------------------------S I M U L A T I O N    L O O P--------------------------------------------------------------------------------
  //----------------------------------------------------------------------------------------------------------------------------------------------------------------------------
  while (framecount < MAX_FRAMES) {

      // Time step
      // float randomFloat = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
      // float c_ambient = randomFloat;    // Ambient value of c
      float c_ambient = C_AMBIENT;

      // if (PERIODIC_FORCE == 1) {
      //     F = Vector2f(magnitude * sin(time), 0.0f); // Initial force
      // }

      // C = Vector2f(dim / 2.0f + 50.0f * sinf(glfwGetTime()), dim / 2.0f);

      // Execute the Navier-Stokes kernel
      NSkernel<<<blocks, threads>>>(dev_u, dev_p, dev_c, dev_obstacleField, c_ambient, gravity, betabouyancy, dx, viscosity, C, F, timestep, r, dim);

      // // Check for CUDA errors
      // hipError_t err = hipGetLastError();
      // if (err != hipSuccess) {
      //     printf("CUDA Error after NSkernel: %s\n", hipGetErrorString(err));
      //     return 1;
      // }

      hipDeviceSynchronize();

      if (framecount % SNAPSHOT_INTERVAL == 0) {
        hipMemcpy(u, dev_u, dim * dim * sizeof(Vector2f),
                   hipMemcpyDeviceToHost);
        vector<float> snapshot;
        snapshot.reserve(dim * dim * 2);
        for (int i = 0; i < dim * dim; ++i)
          snapshot.push_back(u[i].x);
        for (int i = 0; i < dim * dim; ++i)
          snapshot.push_back(u[i].y);
        snapshots.push_back(snapshot);
        // Salva le condizioni al contorno
        left_bc_history.push_back(get_left_bc());
        right_bc_history.push_back(get_right_bc());
        top_bc_history.push_back(get_top_bc());
        bottom_bc_history.push_back(get_bottom_bc());
      }

      framecount++;
      t_current += timestep;
  }
  // --- SCRITTURA SU FILE ---
  if (wantWrite) {
    int M_times = snapshots.size();
    int N_space = snapshots[0].size();

    std::ofstream out(SNAPSHOT_FILENAME);
    std::cout << "Snapshot matrix size (time slices x spatial): " << M_times
              << " x " << N_space << "\n";
    for (int i = 0; i < N_space; ++i) {
      for (int j = 0; j < M_times; ++j) {
        out << snapshots[j][i];
        if (j < M_times - 1)
          out << ",";
      }
      out << "\n";
    }
    out.close();
    printf("Snapshots saved in %s\n", SNAPSHOT_FILENAME);
  }

  // Free memory
  free(u);
  free(p);
  free(c);
  free(obstacleField);
  hipFree(dev_u);
  hipFree(dev_p);
  hipFree(dev_c);
  hipFree(dev_obstacleField);
}

std::vector<std::vector<std::array<float, 5>>> get_left_bc_history() {
  return left_bc_history;
}
std::vector<std::vector<std::array<float, 5>>> get_right_bc_history() {
  return right_bc_history;
}
std::vector<std::vector<std::array<float, 5>>> get_top_bc_history() {
  return top_bc_history;
}
std::vector<std::vector<std::array<float, 5>>> get_bottom_bc_history() {
  return bottom_bc_history;
}

void writeSnapshots(){
    int M_times = snapshots.size();
    int N_space = snapshots[0].size();

    std::ofstream out(SNAPSHOT_FILENAME);
    std::cout << "Snapshot matrix size (time slices x spatial): " << M_times
              << " x " << N_space << "\n";
    for (int i = 0; i < N_space; ++i) {
      for (int j = 0; j < M_times; ++j) {
        out << snapshots[j][i];
        if (j < M_times - 1)
          out << ",";
      }
      out << "\n";
    }
    out.close();
    printf("Snapshots saved in %s\n", SNAPSHOT_FILENAME);
}

// esponi u
std::vector<std::vector<float>> get_u() {
  std::vector<std::vector<float>> u_array(dim, std::vector<float>(dim * 2));
  for (int i = 0; i < dim; ++i) {
    for (int j = 0; j < dim; ++j) {
      int idx = i * dim + j;
      u_array[i][j] = u[idx].x;       // componente x
      u_array[i][j + dim] = u[idx].y; // componente y
    }
  }
  return u_array;
}

std::vector<std::array<float, 4>> get_ic_from_snapshots() {
  std::vector<std::array<float, 4>> ic(dim * dim);
  //==================================================== NOT TESTED=========================================================
  if (snapshots.empty()) {
      printf("Snapshots vector is empty!\n");
      return ic;
  }

  // La prima snapshot contiene [ux(0), ..., ux(N), uy(0), ..., uy(N)]
  const std::vector<float>& first_snapshot = snapshots[0];

  // Safety check
  if (first_snapshot.size() != 2 * dim * dim) {
      printf("First snapshot size mismatch: expected %d, got %zu\n", 2*dim*dim, first_snapshot.size());
      return ic;
  }

  // Griglia
  for (int i = 0; i < dim; ++i) {
      float x = x_min + i * dx;
      for (int j = 0; j < dim; ++j) {
          float y = y_min + j * dy;
          int idx = i * dim + j;
          float ux = first_snapshot[idx];             // primi dim*dim sono ux
          float uy = first_snapshot[idx + dim*dim];   // dopo ci sono gli uy
          ic[idx] = {x, y, ux, uy};
      }
  }
  return ic;
}

py::array_t<float> interpolate_ic_from_snapshots(py::array_t<float, py::array::c_style | py::array::forcecast> query) {
  // query: shape (N, 2) (x, y)
  if (snapshots.empty())
      throw std::runtime_error("Snapshots vector is empty!");
  const std::vector<float>& first_snapshot = snapshots[0];
  if (first_snapshot.size() != 2 * dim * dim)
      throw std::runtime_error("First snapshot size mismatch.");
  printf("shape of first snapshot: %zu\n", first_snapshot.size());
  // float *ux_grid;
  // float* uy_grid ;
  std::vector<float> ux_grid(dim * dim);
  std::vector<float> uy_grid(dim * dim);
  for (int i = 0; i < dim; i++) {
      for (int j = 0; j < dim; j++) {
          int idx = i * dim + j;
          ux_grid[idx] = first_snapshot[idx];             // primi dim*dim sono ux
          uy_grid[idx] = first_snapshot[idx + dim*dim];   // dopo ci sono gli uy
      }
    }
    // float dx = (x_max - x_min) / (dim - 1);
    // float dy = (y_max - y_min) / (dim - 1);

    auto buf = query.request();
  if (buf.ndim != 2 || buf.shape[1] < 2)
      throw std::runtime_error("Input shape must be (N, 2+)");

  int N = buf.shape[0];
  auto ptr = static_cast<const float*>(buf.ptr);

  // Crea output (N, 2)
  py::array_t<float> result({N, 2});
  auto r = result.mutable_unchecked<2>();

  for (int n = 0; n < N; ++n) {
      float x = ptr[n * buf.shape[1]];
      float y = ptr[n * buf.shape[1] + 1];

      int ix = static_cast<int>((x - x_min) / dx);
      int iy = static_cast<int>((y - y_min) / dy);

      ix = std::min(std::max(ix, 0), static_cast<int>(dim) - 2);
      iy = std::min(std::max(iy, 0), static_cast<int>(dim) - 2);


      float x1 = x_min + ix * dx;
      float y1 = y_min + iy * dy;
      float fx = (x - x1) / dx;
      float fy = (y - y1) / dy;

      // int idx11 = ix * dim + iy;
      // int idx12 = ix * dim + (iy + 1);
      // int idx21 = (ix + 1) * dim + iy;
      // int idx22 = (ix + 1) * dim + (iy + 1);
      // x = x_min + ix * dx
// y = y_min + iy * dy
      int idx11 = ix * dim + iy;         // (ix, iy)
      int idx12 = ix * dim + (iy + 1);   // (ix, iy+1)
      int idx21 = (ix + 1) * dim + iy;   // (ix+1, iy)
      int idx22 = (ix + 1) * dim + (iy + 1); // (ix+1, iy+1)


      // int idx11 = iy * dim + ix; // scambia x e y!
      // int idx12 = (ix + 1) * dim + iy;
      // int idx21 = ix * dim + (iy + 1);
      // int idx22 = (iy + 1) * dim + (ix + 1);

      float ux_val = (1-fx)*(1-fy)*ux_grid[idx11]
                   + (1-fx)*fy*ux_grid[idx12]
                   + fx*(1-fy)*ux_grid[idx21]
                   + fx*fy*ux_grid[idx22];
      float uy_val = (1-fx)*(1-fy)*uy_grid[idx11]
                   + (1-fx)*fy*uy_grid[idx12]
                   + fx*(1-fy)*uy_grid[idx21]
                   + fx*fy*uy_grid[idx22];

      r(n, 0) = ux_val;
      r(n, 1) = uy_val;
  }
  return result;
}



// --- PYBIND11 EXPORT ---


PYBIND11_MODULE(navier2d, m) {
  m.def("setupNS2d", &setupNS2d, "Setup CFD 2D");
  m.def("mainNS2d", &mainNS2d, "Setup CFD");
  m.def("get_left_bc", &get_left_bc, "Restituisce il profilo BC su x=0");
  m.def("get_right_bc", &get_right_bc, "Restituisce il profilo BC su x=max");
  m.def("get_top_bc", &get_top_bc, "Restituisce il profilo BC su y=max");
  m.def("get_bottom_bc", &get_bottom_bc, "Restituisce il profilo BC su y=0");
  m.def("get_left_bc_history", &get_left_bc_history,
        "Restituisce la storia delle BC su x=0");
  m.def("get_right_bc_history", &get_right_bc_history,
        "Restituisce la storia delle BC su x=max");
  m.def("get_top_bc_history", &get_top_bc_history,
        "Restituisce la storia delle BC su y=max");
  m.def("get_bottom_bc_history", &get_bottom_bc_history,
        "Restituisce la storia delle BC su y=0");
  m.attr("t_current") = &t_current;
  m.attr("dim") = &dim;
  m.attr("dx") = &dx;
  m.attr("dy") = &dy;
  m.attr("x_min") = &x_min;
  m.attr("x_max") = &x_max;
  m.attr("y_min") = &y_min;
  m.attr("y_max") = &y_max;
  m.def("get_u", &get_u, "Restituisce il campo u come array (dim,dim,2)");
  m.def("get_ic_from_snapshots", &get_ic_from_snapshots, "Restituisce la condizione iniziale completa");
  m.def("interpolate_ic_from_snapshots", &interpolate_ic_from_snapshots, "Interpola le IC CFD sui punti (x,y) richiesti");
  m.def("writeSnapshots", &writeSnapshots, "Scrive gli snapshots su file");
}
